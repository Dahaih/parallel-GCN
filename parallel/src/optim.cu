#include "hip/hip_runtime.h"
#include "../include/optim.cuh"
// #include <cmath>
// #include <cstdlib>

AdamParams AdamParams::get_default() { return {0.001, 0.9, 0.999, 1e-8, 0.0}; }

// ##################################################################################

AdamVariable::AdamVariable(shared_ptr<Variable> var, bool decay_) : dev_data(var->dev_data), dev_grad(var->dev_grad), size(var->size), decay(decay_)
{
    dev_m = dev_shared_ptr<real>(size);
    dev_v = dev_shared_ptr<real>(size);
}

// ##################################################################################

Adam::Adam(std::vector<std::pair<shared_ptr<Variable>, bool>> vars_, AdamParams params_) : params(params_)
{
    step_count = 0;
    for (auto v : vars_)
        vars.emplace_back(v.first, v.second);
    dev_params = dev_shared_ptr<AdamParams>(1);
    dev_params.copy_to_device(&params);
}

__global__ void adam_step_kernel(real *dev_data, real *dev_grad, real *dev_m, real *dev_v, natural size, AdamParams *params, bool decay, real step_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        float grad = dev_grad[i];
        if (decay) // never used, weight decay set to zero
            grad += params->weight_decay * dev_data[i];
        dev_m[i] = params->beta1 * dev_m[i] + (1.0 - params->beta1) * grad;
        dev_v[i] = params->beta2 * dev_v[i] + (1.0 - params->beta2) * grad * grad;
        dev_data[i] -= step_size * dev_m[i] / (sqrtf(dev_v[i]) + params->eps);
    }
}

void Adam::step()
{
    step_count++;
    float step_size = params.lr * sqrtf(1 - powf(params.beta2, step_count)) /
                      (1 - std::pow(params.beta1, step_count));

    for (auto &var : vars)
    {
        dim3 n_blocks(CEIL(var.size, N_THREADS));
        dim3 n_threads(N_THREADS);
        adam_step_kernel<<<n_blocks, n_threads>>>(var.dev_data.get(), var.dev_grad.get(), var.dev_m.get(), var.dev_v.get(), var.size, dev_params.get(), var.decay, step_size);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    }
}

// ##################################################################################