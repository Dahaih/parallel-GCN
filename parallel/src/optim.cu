#include "hip/hip_runtime.h"
#include "../include/optim.cuh"
// #include <cmath>
// #include <cstdlib>

// ##################################################################################

AdamVariable::AdamVariable(shared_ptr<Variable> var, bool decay_, smart_stream &forward_training_stream_) : dev_data(var->dev_data), dev_grad(var->dev_grad), size(var->size), decay(decay_), forward_training_stream(forward_training_stream_)
{
    dev_m = dev_shared_ptr<real>(size);
    dev_v = dev_shared_ptr<real>(size);
    dev_m.set_zero(forward_training_stream);
    dev_v.set_zero(forward_training_stream);
    hipStreamSynchronize(forward_training_stream.get());
}

// ##################################################################################

Adam::Adam(const std::vector<shared_ptr<Variable>> &weights, const std::vector<bool> &decays, AdamParams const *params_, const std::vector<smart_stream> &backward_streams_, std::vector<smart_event> &start_matmul_forward_, smart_stream &forward_training_stream_) : params(params_), step_count(0), backward_streams(backward_streams_), start_matmul_forward(start_matmul_forward_), forward_training_stream(forward_training_stream_)
{
    if (weights.size() != decays.size())
    {
        std::cout << "Error in Adam constructor: weights and decays must have the same size" << std::endl;
        exit(1);
    }

    for (natural i = 0; i < weights.size(); i++)
        vars.emplace_back(weights[i], decays[i], forward_training_stream);

    weight_decay = dev_shared_ptr<real>(1);
    beta1 = dev_shared_ptr<real>(1);
    beta2 = dev_shared_ptr<real>(1);
    eps = dev_shared_ptr<real>(1);
    weight_decay.copy_to_device_async(&(params->weight_decay), forward_training_stream);
    beta1.copy_to_device_async(&(params->beta1), forward_training_stream);
    beta2.copy_to_device_async(&(params->beta2), forward_training_stream);
    eps.copy_to_device_async(&(params->eps), forward_training_stream);
    hipStreamSynchronize(forward_training_stream.get());
}

// ##################################################################################

__global__ void adam_step_kernel(real *dev_data, const real *dev_grad, real *dev_m, real *dev_v, const natural size, const real *weight_decay, const real *beta1, const real *beta2, const real *eps, const bool decay, const real step_size)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < size; i += blockDim.x * gridDim.x)
    {
        real grad = dev_grad[i];
        if (decay)
            grad += (*weight_decay) * dev_data[i];
        dev_m[i] = (*beta1) * dev_m[i] + (1.0 - (*beta1)) * grad;
        dev_v[i] = (*beta2) * dev_v[i] + (1.0 - (*beta2)) * grad * grad;
        dev_data[i] -= step_size * dev_m[i] / (sqrtf(dev_v[i]) + (*eps));
    }
}

void Adam::step()
{
    // timer_start(TMR_OPTIMIZER);

    step_count++;
    const real step_size = params->learning_rate * sqrtf(1 - powf(params->beta2, step_count)) / (1 - powf(params->beta1, step_count));
    // hipStreamSynchronize(forward_training_stream.get());
    // hipStreamSynchronize(streams[1].get());
    /*
    for (const auto &var : vars)
    {
        const natural n_blocks = std::min(CEIL(var.size, CudaParams::N_THREADS), static_cast<natural>(N_BLOCKS));
        adam_step_kernel<<<n_blocks, CudaParams::N_THREADS, 0, forward_training_stream.get()>>>(var.dev_data.get(), var.dev_grad.get(), var.dev_m.get(), var.dev_v.get(), var.size, weight_decay.get(), beta1.get(), beta2.get(), eps.get(), var.decay, step_size);
#ifdef DEBUG_CUDA
        CHECK_CUDA_ERROR(hipGetLastError());
#endif
    }
    */
    natural i = 0;
    const natural n_blocks = std::min(CEIL(vars[i].size, CudaParams::N_THREADS), static_cast<natural>(CudaParams::N_BLOCKS));
    adam_step_kernel<<<n_blocks, CudaParams::N_THREADS, 0, backward_streams[i].get()>>>(vars[i].dev_data.get(), vars[i].dev_grad.get(), vars[i].dev_m.get(), vars[i].dev_v.get(), vars[i].size, weight_decay.get(), beta1.get(), beta2.get(), eps.get(), vars[i].decay, step_size);
    hipEventRecord(start_matmul_forward[i].get(), backward_streams[i].get());
#ifdef DEBUG_CUDA
    CHECK_CUDA_ERROR(hipGetLastError());
#endif
    i++;

    for (; i < vars.size(); i++)
    {
        const natural n_blocks = std::min(CEIL(vars[i].size, CudaParams::N_THREADS), static_cast<natural>(CudaParams::N_BLOCKS));
        adam_step_kernel<<<n_blocks, CudaParams::N_THREADS, 0, backward_streams[1].get()>>>(vars[i].dev_data.get(), vars[i].dev_grad.get(), vars[i].dev_m.get(), vars[i].dev_v.get(), vars[i].size, weight_decay.get(), beta1.get(), beta2.get(), eps.get(), vars[i].decay, step_size);
        hipEventRecord(start_matmul_forward[i].get(), backward_streams[1].get());
#ifdef DEBUG_CUDA
        CHECK_CUDA_ERROR(hipGetLastError());
#endif
    }

    // timer_stop(TMR_OPTIMIZER);
}

// ##################################################################################