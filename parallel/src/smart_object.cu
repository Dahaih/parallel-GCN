#include "../include/smart_object.cuh"

template <>
smart_object<hipStream_t>::smart_object() : refCount(new size_t(1))
{
    hipStreamCreate(&object);
}

template <>
smart_object<hipEvent_t>::smart_object() : refCount(new size_t(1))
{
    hipEventCreate(&object);
}

template <>
void smart_object<hipStream_t>::DecrementRefCount()
{
    if (refCount && --(*refCount) == 0)
    {
        delete refCount;
        if (object != nullptr)
            hipStreamDestroy(object);
    }
}

template <>
void smart_object<hipEvent_t>::DecrementRefCount()
{
    if (refCount && --(*refCount) == 0)
    {
        delete refCount;
        if (object != nullptr)
            hipEventDestroy(object);
    }
}