#include "hip/hip_runtime.h"
#include "../include/module.cuh"

// DROPOUT
// ##################################################################################

Dropout::Dropout(shared_ptr<Variable> in_, real p_, dev_shared_ptr<randState> dev_rand_states_) : in(in_), p(p_), dev_rand_states(dev_rand_states_)
{
    if (in->dev_grad.get())
        dev_mask = dev_shared_ptr<bool>(in->size);
    else
        dev_mask = dev_shared_ptr<bool>();
}

// ##################################################################################
/*
__global__ void dropout_kernel_forward(real *dev_data, bool *dev_mask, randState *dev_rand_states,
                                       const natural size, const real p, const real scale)
{
    __shared__ randState s_rand_states[N_THREADS_DROPOUT];
    s_rand_states[threadIdx.x] = dev_rand_states[threadIdx.x];
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < size; i += blockDim.x * gridDim.x)
    {
        bool keep = hiprand_uniform(&s_rand_states[threadIdx.x]) >= p;
        dev_data[i] *= keep ? scale : 0.f;
        if (dev_mask)
            dev_mask[i] = keep;
    }
}
*/
// needs hiprandStatePhilox4_32_10_t
__global__ void dropout_kernel_forward(real *dev_data, bool *dev_mask, randState *dev_rand_states,
                                       const natural size, const real p, const real scale)
{
    __shared__ randState s_rand_states[N_THREADS_DROPOUT];
    s_rand_states[threadIdx.x] = dev_rand_states[threadIdx.x];
    natural id = 4 * (blockIdx.x * blockDim.x + threadIdx.x);

#pragma unroll
    for (natural i = id; i < size; i += 4 * blockDim.x * gridDim.x)
    {
        float4 rand = hiprand_uniform4(&s_rand_states[threadIdx.x]);
        dev_data[i] *= rand.x >= p ? scale : 0;
        dev_data[i + 1] *= rand.y >= p ? scale : 0;
        dev_data[i + 2] *= rand.z >= p ? scale : 0;
        dev_data[i + 3] *= rand.w >= p ? scale : 0;
        if (dev_mask)
        {
            dev_mask[i] = rand.x >= p;
            dev_mask[i + 1] = rand.y >= p;
            dev_mask[i + 2] = rand.z >= p;
            dev_mask[i + 3] = rand.w >= p;
        }
    }
}

void Dropout::forward(bool training)
{
    if (!training)
        return;
    timer_start(TMR_DROPOUT_FW);
    real scale = 1.0 / (1.0 - p);
    natural n_blocks = std::min(CEIL(in->size, N_THREADS_DROPOUT), static_cast<natural>(N_BLOCKS));
    // natural n_blocks = CEIL(in->size, N_THREADS_DROPOUT);
    dropout_kernel_forward<<<n_blocks, N_THREADS_DROPOUT>>>(in->dev_data.get(), dev_mask.get(), dev_rand_states.get(), in->size, p, scale);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    // std::cout << "size of rand: " << sizeof(hiprandStatePhilox4_32_10_t) << std::endl;
    // std::cout << "n_blocks: " << n_blocks << std::endl;
    timer_stop(TMR_DROPOUT_FW);
}

// ##################################################################################

__global__ void dropout_kernel_backward(real *dev_grad, const bool *mask, const natural size, const real scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (natural i = id; i < size; i += blockDim.x * gridDim.x)
    {
        dev_grad[i] *= mask[i] ? scale : 0.f;
    }
}

void Dropout::backward()
{
    if (!dev_mask.get())
        return;
    timer_start(TMR_DROPOUT_BW);

    const real scale = 1.0 / (1.0 - p);
    natural n_blocks = std::min(CEIL(in->size, N_THREADS), static_cast<natural>(N_BLOCKS));
    dropout_kernel_backward<<<n_blocks, N_THREADS>>>(in->dev_grad.get(), dev_mask.get(), in->size, scale);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_DROPOUT_BW);
}

// ##################################################################################

// SPARSEMATMUL
// ##################################################################################

SparseMatmul::SparseMatmul(shared_ptr<Variable> a_, shared_ptr<Variable> b_, shared_ptr<Variable> c_, DevSparseIndex *sp_, natural m_, natural n_, natural p_) : a(a_), b(b_), c(c_), sp(sp_), m(m_), n(n_), p(p_) {}

// ##################################################################################

__global__ void sparse_matmul_kernel_forward(const real *a, const real *b, real *c, const natural *indptr, const natural *indices, natural m, natural p)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < m * p; i += blockDim.x * gridDim.x)
    {
        natural row = i / p;
        natural col = i % p;
        if (row >= m || col >= p)
            return;
        real sum = 0;
#pragma unroll
        for (natural jj = indptr[row]; jj < indptr[row + 1]; jj++)
        {
            natural j = indices[jj];
            sum += a[jj] * b[j * p + col];
        }
        c[i] = sum;
    }
}

void SparseMatmul::forward(bool training)
{
    timer_start(TMR_SPMATMUL_FW);

    natural n_blocks = std::min(CEIL(m * p, N_THREADS), static_cast<natural>(N_BLOCKS));
    sparse_matmul_kernel_forward<<<n_blocks, N_THREADS>>>(a->dev_data.get(), b->dev_data.get(), c->dev_data.get(), sp->dev_indptr.get(), sp->dev_indices.get(), m, p);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_SPMATMUL_FW);
}

// ##################################################################################

__global__ void sparse_matmul_kernel_backward(const real *a, real *b, const real *c, const natural *indptr, const natural *indices, const natural m, const natural p)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < m * p; i += blockDim.x * gridDim.x)
    {
        natural row = i / p;
        natural col = i % p;
        if (row >= m || col >= p)
            return;
#pragma unroll
        for (natural jj = indptr[row]; jj < indptr[row + 1]; jj++)
        {
            natural j = indices[jj];
            atomicAdd(&b[j * p + col], a[jj] * c[row * p + col]);
            // b[j * p + col] += a[jj] * c[row * p + col]; BUG!
        }
    }
}

void SparseMatmul::backward()
{
    timer_start(TMR_SPMATMUL_BW);

    b->zero_grad();
    natural n_blocks = std::min(CEIL(m * p, N_THREADS), static_cast<natural>(N_BLOCKS));
    sparse_matmul_kernel_backward<<<n_blocks, N_THREADS>>>(a->dev_data.get(), b->dev_grad.get(), c->dev_grad.get(), sp->dev_indptr.get(), sp->dev_indices.get(), m, p);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_SPMATMUL_BW);
}

// serial version
/*
void SparseMatmul::backward()
{
    timer_start(TMR_SPMATMUL_BW);
    std::vector<real> my_a(a->size);
    std::vector<real> my_c_grad(c->size);
    std::vector<real> my_b_grad(b->size, 0.f);
    std::vector<natural> my_indptr(sp->indptr_size);
    std::vector<natural> my_indices(sp->indices_size);
    a->dev_data.copy_to_host(my_a.data());
    c->dev_grad.copy_to_host(my_c_grad.data());

    sp->dev_indptr.copy_to_host(my_indptr.data());
    sp->dev_indices.copy_to_host(my_indices.data());

    b->zero_grad();
    int row = 0;
    for (int i = 0; i < my_indptr.size() - 1; i++)
        for (int k = 0; k < p; k++)
            for (int jj = my_indptr[i]; jj < my_indptr[i + 1]; jj++)
            {
                int j = my_indices[jj];

                my_b_grad[j * p + k] += my_c_grad[i * p + k] * my_a[jj];
            }
    b->dev_grad.copy_to_device(my_b_grad.data());
    timer_stop(TMR_SPMATMUL_BW);
}
*/
/*
__global__ void cuda_SparseMatmul_backward_kernel(float *a_in, float *b_in, float *c_in, natural *indptr, natural *indices, int p)
{
    int i = blockIdx.x;
    int k = threadIdx.x;

#pragma unroll
    for (int jj = indptr[i]; jj < indptr[i + 1]; jj++)
    {
        int j = indices[jj];
        b_in[j * p + k] += c_in[i * p + k] * a_in[jj];
    }
}

void SparseMatmul::backward()
{
    timer_start(TMR_SPMATMUL_BW);

    b->zero_grad();
    // TODO: when p larger than 1024?
    if (sp->indptr_size <= 1)
        return;
    dim3 block(sp->indptr_size - 1, 1, 1);
    dim3 thread_in_block(p, 1, 1);
    cuda_SparseMatmul_backward_kernel<<<block, thread_in_block>>>(a->dev_data.get(), b->dev_grad.get(), c->dev_grad.get(), sp->dev_indptr.get(), sp->dev_indices.get(), p);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_SPMATMUL_BW);
}
*/
// GRAPHSUM
// ##################################################################################

GraphSum::GraphSum(shared_ptr<Variable> in_, shared_ptr<Variable> out_, DevSparseIndex *graph_, dev_shared_ptr<real> dev_graph_value_, natural dim_) : in(in_), out(out_), graph(graph_), dev_graph_value(dev_graph_value_), dim(dim_) {}

// ##################################################################################

// n è H mentre m è N
__global__ void graphsum_kernel(const real *dev_graph_value, const real *dev_in, real *dev_out, const natural *indptr, const natural *indices, const natural m, const natural n)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < m * n; i += blockDim.x * gridDim.x)
    {
        natural row = i / n;
        natural col = i % n;
        if (row >= m || col >= n)
            return;
        real sum = 0;
#pragma unroll
        for (natural jj = indptr[row]; jj < indptr[row + 1]; jj++)
        {
            natural j = indices[jj];
            // float coef = 1.0 / sqrtf((indptr[row + 1] - indptr[row]) * (indptr[j + 1] - indptr[j]));
            sum += dev_graph_value[jj] * dev_in[j * n + col];
        }
        dev_out[i] = sum;
    }
}

void GraphSum::forward(bool training)
{

    timer_start(TMR_GRAPHSUM_FW);

    const natural numNodes = graph->indptr_size - 1;
    const natural n_blocks = std::min(CEIL(numNodes * dim, N_THREADS), static_cast<natural>(N_BLOCKS));
    graphsum_kernel<<<n_blocks, N_THREADS>>>(dev_graph_value.get(), in->dev_data.get(), out->dev_data.get(), graph->dev_indptr.get(), graph->dev_indices.get(), numNodes, dim);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_GRAPHSUM_FW);
}

// ###############################################################################

void GraphSum::backward()
{
    timer_start(TMR_GRAPHSUM_BW);

    const natural numNodes = graph->indptr_size - 1;
    const natural n_blocks = std::min(CEIL(numNodes * dim, N_THREADS), static_cast<natural>(N_BLOCKS));
    graphsum_kernel<<<n_blocks, N_THREADS>>>(dev_graph_value.get(), out->dev_grad.get(), in->dev_grad.get(), graph->dev_indptr.get(), graph->dev_indices.get(), numNodes, dim);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_GRAPHSUM_BW);
}

// RELU
// ##################################################################################

ReLU::ReLU(shared_ptr<Variable> in_) : in(in_)
{
    dev_mask = dev_shared_ptr<bool>(in->size);
}

// ##################################################################################

__global__ void relu_kernel_forward(real *dev_data, bool *dev_mask, const natural size, const bool training)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < size; i += blockDim.x * gridDim.x)
    {
        bool keep = dev_data[i] > 0;
        if (training)
            dev_mask[i] = keep;
        if (!keep)
            dev_data[i] = 0.f;
    }
}

void ReLU::forward(bool training)
{
    timer_start(TMR_RELU_FW);

    const natural n_blocks = std::min(CEIL(in->size, N_THREADS), static_cast<natural>(N_BLOCKS));
    relu_kernel_forward<<<n_blocks, N_THREADS>>>(in->dev_data.get(), dev_mask.get(), in->size, training);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_RELU_FW);
}

// ##################################################################################

__global__ void relu_kernel_backward(real *d_in_grad, const bool *d_mask, const natural size)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < size; i += blockDim.x * gridDim.x)
    {
        if (!d_mask[i])
            d_in_grad[i] = 0.f;
    }
}

void ReLU::backward()
{
    timer_start(TMR_RELU_BW);

    const natural n_blocks = std::min(CEIL(in->size, N_THREADS), static_cast<natural>(N_BLOCKS));
    relu_kernel_backward<<<n_blocks, N_THREADS>>>(in->dev_grad.get(), dev_mask.get(), in->size);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_RELU_BW);
}

// MATMUL
// ##################################################################################

Matmul::Matmul(shared_ptr<Variable> a_, shared_ptr<Variable> b_, shared_ptr<Variable> c_, natural m_, natural n_, natural p_) : a(a_), b(b_), c(c_), m(m_), n(n_), p(p_) {}

// ##################################################################################

// esperimenti con tiles rettangolari
/*
__global__ void matmul_kernel(const real *a, const real *b, real *c, const natural m, const natural n, const natural p)
{
    // shared memory arrays that are used as tiles to store a portion of matrices A and B.
    __shared__ real tile_a[TILE_DIM_Y][TILE_DIM_X];
    __shared__ real tile_b[TILE_DIM_Y][TILE_DIM_X];
    natural tx = threadIdx.x;
    natural ty = threadIdx.y;
    // row/col index of the current element of the result matrix that is computed by the thread.
    natural row = blockIdx.y * TILE_DIM_Y + ty;
    natural col = blockIdx.x * TILE_DIM_X + tx;
    // number of tile rows/columns needed to cover the matrices A and B
    natural range_x = CEIL(n, TILE_DIM_X);
    //  partial sum of the result matrix element computed by the thread
    real res = 0;

#pragma unroll
    // iterates over the tiles needed to compute the result matrix element
    for (natural i = 0; i < range_x; i++)
    {
        // check if the current thread is within the boundaries of A .
        if (row < m && i * TILE_DIM_X + tx < n)
            // load a portion of matrix A into the shared memory tiles.
            tile_a[ty][tx] = a[row * n + i * TILE_DIM_X + tx];
        else
            tile_a[ty][tx] = 0;
        // check if the current thread is within the boundaries of  B.
        if (col < p && i * TILE_DIM_Y + ty < n)
            // load a portion of matrix B into the shared memory tiles.
            tile_b[ty][tx] = b[(i * TILE_DIM_Y + ty) * p + col];
        else
            tile_b[ty][tx] = 0;
        // synchronizes all threads in the block before executing the next set of instructions.
        __syncthreads();
#pragma unroll
        // computes the partial sum of the result matrix element using the shared memory tiles
        for (natural j = 0; j < TILE_DIM_X; j++)
            res += tile_a[ty][j] * tile_b[j][tx];
        __syncthreads();
    }
    // stores the result of the partial sum in the result matrix if the thread is within the boundaries of the result matrix
    if (row < m && col < p)
        c[row * p + col] = res;
}

void Matmul::forward(bool training)
{
    timer_start(TMR_MATMUL_FW);
    // c->zero();
    dim3 n_blocks(CEIL(p, TILE_DIM_X), CEIL(m, TILE_DIM_Y));
    std::cout << "n_blocks: " << n_blocks.x << " " << n_blocks.y << std::endl;
    dim3 n_threads(TILE_DIM_X, TILE_DIM_Y);

    matmul_kernel<<<n_blocks, n_threads>>>(a->dev_data.get(), b->dev_data.get(), c->dev_data.get(), m, n, p);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_MATMUL_FW);
}
*/

__global__ void matmul_kernel_forward(const real *a, const real *b, real *c, const natural m, const natural n, const natural p)
{
    // shared memory arrays that are used as tiles to store a portion of matrices A and B.
    __shared__ real tile_a[TILE_DIM][TILE_DIM];
    __shared__ real tile_b[TILE_DIM][TILE_DIM];
    natural tx = threadIdx.x;
    natural ty = threadIdx.y;
#pragma unroll
    for (natural row = blockIdx.y * TILE_DIM + ty; row < m; row += blockDim.y * gridDim.y)
    {
        natural col = blockIdx.x * TILE_DIM + tx;
        //  number of tile rows/columns needed to cover the matrices A and B
        natural range = CEIL(n, TILE_DIM);
        //  partial sum of the result matrix element computed by the thread
        real res = 0;

#pragma unroll
        // iterates over the tiles needed to compute the result matrix element
        for (natural i = 0; i < range; i++)
        {
            // check if the current thread is within the boundaries of A .
            if (i * TILE_DIM + tx < n)
                // load a portion of matrix A into the shared memory tiles.
                tile_a[ty][tx] = a[row * n + i * TILE_DIM + tx];
            else
                tile_a[ty][tx] = 0;
            // check if the current thread is within the boundaries of  B.
            if (col < p && i * TILE_DIM + ty < n)
                // load a portion of matrix B into the shared memory tiles.
                tile_b[ty][tx] = b[(i * TILE_DIM + ty) * p + col];

            else
                tile_b[ty][tx] = 0;
            // synchronizes all threads in the block before executing the next set of instructions.
            __syncthreads();
#pragma unroll
            // computes the partial sum of the result matrix element using the shared memory tiles
            for (natural j = 0; j < TILE_DIM; j++)
                res += tile_a[ty][j] * tile_b[j][tx];

            __syncthreads();
        }
        // stores the result of the partial sum in the result matrix if the thread is within the boundaries of the result matrix
        if (col < p)
            c[row * p + col] = res;
    }
}

void Matmul::forward(bool training)
{
    timer_start(TMR_MATMUL_FW);
    // c->zero();
    const natural n_blocks_y = std::min(CEIL(m, TILE_DIM), static_cast<natural>(N_BLOCKS));
    dim3 n_blocks(CEIL(p, TILE_DIM), n_blocks_y);
    // std::cout << "n_blocks: " << n_blocks.x << " " << n_blocks.y << std::endl;
    dim3 n_threads(TILE_DIM, TILE_DIM);

    matmul_kernel_forward<<<n_blocks, n_threads>>>(a->dev_data.get(), b->dev_data.get(), c->dev_data.get(), m, n, p);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_MATMUL_FW);
}

// ##################################################################################

// ###### versione tutto in un kernel (lenta)
/*
__global__ void matmul_kernel_backward_1(real *a_grad, real *b_grad, const real *a_data, const real *b_data, const real *c_grad, const natural m, const natural n, const natural p)
{
    // shared memory arrays that are used as tiles to store a portion of matrices A and B.
    __shared__ real tile_c_grad[TILE_DIM][TILE_DIM];
    __shared__ real tile_b_data[TILE_DIM][TILE_DIM];
    natural tx = threadIdx.x;
    natural ty = threadIdx.y;
#pragma unroll
    for (natural row = blockIdx.y * TILE_DIM + ty; row < m; row += blockDim.y * gridDim.y)
    {
        natural col = blockIdx.x * TILE_DIM + tx;
        //  number of tile rows/columns needed to cover the matrices A and B
        natural range = CEIL(p, TILE_DIM);
        //  partial sum of the result matrix element computed by the thread
        real res = 0;

#pragma unroll
        // iterates over the tiles needed to compute the result matrix element
        for (natural i = 0; i < range; i++)
        {
            // check if the current thread is within the boundaries of C .
            if (i * TILE_DIM + tx < p)
                // load a portion of matrix A into the shared memory tiles.
                tile_c_grad[ty][tx] = c_grad[row * p + i * TILE_DIM + tx];
            else
                tile_c_grad[ty][tx] = 0;
            // check if the current thread is within the boundaries of  B.
            if (col < n && i * TILE_DIM + ty < p)
                // load a portion of matrix B into the shared memory tiles.
                // tile_b[ty][tx] = b[(i * TILE_DIM + ty) * p + col];
                tile_b_data[ty][tx] = b_data[col * p + i * TILE_DIM + ty];
            else
                tile_b_data[ty][tx] = 0;
            // synchronizes all threads in the block before executing the next set of instructions.
            __syncthreads();
#pragma unroll
            // computes the partial sum of the result matrix element using the shared memory tiles
            for (natural k = 0; k < TILE_DIM; k++)
            {
                // res += tile_c_grad[ty][j] * tile_b[tx][j];
                res += tile_c_grad[ty][k] * tile_b_data[k][tx];
                b_grad[col * p + k] += a_data[row * n + col] * tile_c_grad[ty][k];
            }
            __syncthreads();
        }
        // stores the result of the partial sum in the result matrix if the thread is within the boundaries of the result matrix
        if (col < n)
            a_grad[row * n + col] = res;
    }
}

void Matmul::backward()
{
    timer_start(TMR_MATMUL_BW);

    b->zero_grad();
    const natural n_blocks_y_1 = std::min(CEIL(m, TILE_DIM), static_cast<natural>(N_BLOCKS));
    dim3 n_blocks_1(CEIL(n, TILE_DIM), n_blocks_y_1);

    dim3 n_threads(TILE_DIM, TILE_DIM);

    dim3 n_blocks_2((p - 1) / TILE_DIM + 1, (n - 1) / TILE_DIM + 1, 1);

    // matmul_kernel_backward_1<<<n_blocks_1, n_threads>>>(a->dev_grad.get(), b->dev_data.get(), c->dev_grad.get(), m, n, p);
    matmul_kernel_backward_1<<<n_blocks_1, n_threads>>>(a->dev_grad.get(), b->dev_grad.get(), a->dev_data.get(), b->dev_data.get(), c->dev_grad.get(), m, n, p);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    // matmul_kernel_backward_2<<<n_blocks_1, n_threads>>>(b->dev_grad.get(), a->dev_data.get(), c->dev_grad.get(), m, n, p);
    // CHECK_CUDA_ERROR(hipGetLastError());
    // CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_MATMUL_BW);
}
*/

__global__ void matmul_kernel_backward_1(real *a, const real *b, const real *c, const natural m, const natural n, const natural p)
{
    // shared memory arrays that are used as tiles to store a portion of matrices A and B.
    __shared__ real tile_c[TILE_DIM][TILE_DIM];
    __shared__ real tile_b[TILE_DIM][TILE_DIM];
    natural tx = threadIdx.x;
    natural ty = threadIdx.y;
#pragma unroll
    for (natural row = blockIdx.y * TILE_DIM + ty; row < m; row += blockDim.y * gridDim.y)
    {
        natural col = blockIdx.x * TILE_DIM + tx;
        //  number of tile rows/columns needed to cover the matrices A and B
        natural range = CEIL(p, TILE_DIM);
        //  partial sum of the result matrix element computed by the thread
        real res = 0;

#pragma unroll
        // iterates over the tiles needed to compute the result matrix element
        for (natural i = 0; i < range; i++)
        {
            // check if the current thread is within the boundaries of C .
            if (i * TILE_DIM + tx < p)
                // load a portion of matrix A into the shared memory tiles.
                tile_c[ty][tx] = c[row * p + i * TILE_DIM + tx];
            else
                tile_c[ty][tx] = 0;
            // check if the current thread is within the boundaries of  B.
            if (col < n && i * TILE_DIM + ty < p)
                // load a portion of matrix B into the shared memory tiles.
                tile_b[ty][tx] = b[col * p + i * TILE_DIM + ty];
            else
                tile_b[ty][tx] = 0;
            // synchronizes all threads in the block before executing the next set of instructions.
            __syncthreads();
#pragma unroll
            // computes the partial sum of the result matrix element using the shared memory tiles
            for (natural k = 0; k < TILE_DIM; k++)
                res += tile_c[ty][k] * tile_b[k][tx];

            __syncthreads();
        }
        // stores the result of the partial sum in the result matrix if the thread is within the boundaries of the result matrix
        if (col < n)
            a[row * n + col] = res;
    }
}

__global__ void matmul_kernel_backward_2(const real *a, real *b, const real *c, const natural m, const natural n, const natural p)
{
    // shared memory arrays that are used as tiles to store a portion of matrices A and B.
    __shared__ real tile_a[TILE_DIM][TILE_DIM];
    __shared__ real tile_c[TILE_DIM][TILE_DIM];
    natural tx = threadIdx.x;
    natural ty = threadIdx.y;

    natural row = blockIdx.y * TILE_DIM + ty;
    natural col = blockIdx.x * TILE_DIM + tx;
    natural range = CEIL(m, TILE_DIM);
    real res = 0;
    // iterates over the tiles needed to compute the result matrix element
#pragma unroll
    for (natural i = 0; i < range; i++)
    {
        if (row < n && i * TILE_DIM + tx < m)
            tile_a[ty][tx] = a[(i * TILE_DIM + tx) * n + row];
        else
            tile_a[ty][tx] = 0;
        if (col < p && i * TILE_DIM + ty < m)
            tile_c[ty][tx] = c[(i * TILE_DIM + ty) * p + col];
        else
            tile_c[ty][tx] = 0;
        __syncthreads();

#pragma unroll
        for (natural k = 0; k < TILE_DIM; k++)
            res += tile_a[ty][k] * tile_c[k][tx];
        __syncthreads();
    }

    if (row < n && col < p)
        b[row * p + col] = res;
}

void Matmul::backward()
{
    timer_start(TMR_MATMUL_BW);

    // b->zero_grad();
    // a->zero_grad();
    const natural n_blocks_y_1 = std::min(CEIL(m, TILE_DIM), static_cast<natural>(N_BLOCKS));
    dim3 n_blocks_1(CEIL(n, TILE_DIM), n_blocks_y_1);
    dim3 n_blocks_2(CEIL(p, TILE_DIM), CEIL(n, TILE_DIM));
    dim3 n_threads(TILE_DIM, TILE_DIM);
    matmul_kernel_backward_1<<<n_blocks_1, n_threads>>>(a->dev_grad.get(), b->dev_data.get(), c->dev_grad.get(), m, n, p);
    CHECK_CUDA_ERROR(hipGetLastError());
    matmul_kernel_backward_2<<<n_blocks_2, n_threads>>>(a->dev_data.get(), b->dev_grad.get(), c->dev_grad.get(), m, n, p);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    timer_stop(TMR_MATMUL_BW);
}

// CROSSENTROPYLOSS
// ##################################################################################

CrossEntropyLoss::CrossEntropyLoss(shared_ptr<Variable> logits_, dev_shared_ptr<integer> dev_truth_, real *loss_, natural num_classes_) : logits(logits_), dev_truth(dev_truth_), loss(loss_), num_classes(num_classes_)
{
    dev_loss = dev_shared_ptr<real>(logits->size / num_classes); // N elements (number of nodes)
    dev_loss_res = dev_shared_ptr<real>(1);
}

// ##################################################################################

__global__ void cross_entropy_loss_kernel(real *dev_data, real *dev_grad, const integer *dev_truth, real *dev_loss, const natural num_classes, const natural num_nodes, const natural num_samples, bool training)
{
    natural id = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
    for (natural i = id; i < num_nodes; i += blockDim.x * gridDim.x)
    {
        if (dev_truth[i] < 0)
            return;
        real *logit = &dev_data[i * num_classes];
        real sum_exp = 0.;
        real max_logit = logit[0];
#pragma unroll
        // get the maximum value of each node
        for (natural j = 1; j < num_classes; j++)
            max_logit = fmax(max_logit, logit[j]);
#pragma unroll
        for (natural j = 0; j < num_classes; j++)
        {
            logit[j] -= max_logit; // numerical stability
            sum_exp += expf(logit[j]);
        }
        dev_loss[i] = logf(sum_exp) - logit[dev_truth[i]];
        if (training)
        {
#pragma unroll
            for (natural j = 0; j < num_classes; j++)
            {
                real prob = expf(logit[j]) / sum_exp;
                dev_grad[i * num_classes + j] = prob / num_samples;
            }
            dev_grad[i * num_classes + dev_truth[i]] -= 1.0 / num_samples;
        }
    }
}

void CrossEntropyLoss::forward(bool training)
{

    timer_start(TMR_LOSS_FW);

    if (training)
        logits->zero_grad();

    dev_loss.set_zero();
    const natural DIM = logits->size / num_classes;
    // print_gpu<real>(dev_loss, DIM, DIM);
    // print_gpu<integer>(dev_truth, DIM, DIM);
    const natural n_blocks = std::min(CEIL(DIM, N_THREADS), static_cast<natural>(N_BLOCKS));
    cross_entropy_loss_kernel<<<n_blocks, N_THREADS>>>(logits->dev_data.get(), logits->dev_grad.get(), dev_truth.get(), dev_loss.get(), num_classes, DIM, num_samples, training);
    // cross_entropy_loss_kernel<<<DIM, 1>>>(logits->dev_data.get(), logits->dev_grad.get(), dev_truth.get(), dev_loss.get(), num_classes, num_samples, training);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    // print_gpu<real>(dev_loss, DIM, DIM);

    // METODO 1 gpu

    dev_loss_res.set_zero();
    reduce<<<n_blocks, N_THREADS>>>(dev_loss.get(), dev_loss_res.get(), DIM);
    *loss = 0;
    dev_loss_res.copy_to_host(loss);
    *loss /= num_samples;

    // METODO 2 cpu
    /*
    std::vector<real> loss_cpu(DIM);
    dev_loss.copy_to_host(loss_cpu.data());
    // print_cpu(loss_cpu, DIM);
    *loss = std::accumulate(loss_cpu.begin(), loss_cpu.end(), static_cast<real>(0)) / num_samples;
    // std::cout << "loss: " << std::accumulate(loss_cpu.begin(), loss_cpu.end(), static_cast<real>(0)) / num_samples << std::endl;
*/
    // METODO 3 reduce_gpu()
    // *loss = reduce_gpu(dev_loss.get(), DIM) / num_samples;

    timer_stop(TMR_LOSS_FW);
}

void CrossEntropyLoss::backward()
{
}

void CrossEntropyLoss::set_num_samples(natural num_samples_)
{
    num_samples = num_samples_;
}

natural CrossEntropyLoss::get_num_samples() const
{
    return num_samples;
};
